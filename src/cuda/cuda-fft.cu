// TUBES SISTER 13520002 CUDA
// cuda-fft.cu
 
// how to run
// > nvcc cuda-fft.cu -o cuda-fft
// > ./cuda-fft
 
// how to measure time
// ex testcase.txt already created, create empty output.txt
// > time ./cuda-fft < 128.txt > output.txt


#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define MAX_N 512
#define BLOCK_SIZE 16
#define CU_MPI make_hipDoubleComplex(M_PI, 0.0)

struct Matrix {
    int    size;
    double mat[MAX_N*MAX_N];
};

struct FreqMatrix {
    int    size;
    hipDoubleComplex mat[MAX_N*MAX_N];
};

void readMatrix(struct Matrix *m) {
    scanf("%d", &(m->size));
    for (int i = 0; i < m->size; i++)
        for (int j = 0; j < m->size; j++)
            scanf("%lf", &(m->mat[i*m->size+j]));
}

__host__ __device__ hipDoubleComplex _cuCexp (hipDoubleComplex arg)
{
   hipDoubleComplex res;
   double s, c;
   double e = exp(arg.x);
   sincos(arg.y, &s, &c);
   res.x = c * e;
   res.y = s * e;
   return res;
}

__global__ void fft(double *d_mat, hipDoubleComplex *d_freq, int size) {
    // get index of thread
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int l = blockIdx.y * blockDim.y + threadIdx.y;

    if (k < size && l < size) {
        // Rumus 2D FFT
        // F[k,l] = 1/MN * { 
        // sum(sum(f[m,n] * e^((-2*pi*i) * (k*m/M + l*n/N))) +                              for calculate even row and even column
        // sum(sum(f[m,n] * e^((-2*pi*i) * (k*m/M + l*n/N)) * e^((-2*pi*i) * (l/N))) +      for calculate even row and odd column
        // sum(sum(f[m,n] * e^((-2*pi*i) * (k*m/M + l*n/N)) * e^((-2*pi*i) * (k/M))) +      for calculate odd row and even column
        // sum(sum(f[m,n] * e^((-2*pi*i) * (k*m/M + l*n/N)) * e^((-2*pi*i) * ((l+k)/M)))    for calculate odd row and odd column
        // }

        // make size to cuDoubleComplex
        hipDoubleComplex sizeSquare      = make_hipDoubleComplex(size*size, 0.0);
        // initialize element in cuDoubleComplex format = 0 + 0i
        hipDoubleComplex element         = make_hipDoubleComplex(0.0, 0.0);  
        // make -2 to cuDoubleComplex
        hipDoubleComplex var_exp         = make_hipDoubleComplex(0.0, -2.0);

        // arg for even row and even column
        // 0
        // arg for even row and odd column
        hipDoubleComplex arg_even_odd    = make_hipDoubleComplex(l / (double) size, 0.0);
        // arg for odd row and even column
        hipDoubleComplex arg_odd_even    = make_hipDoubleComplex(k / (double) size, 0.0);
        // arg for odd row and odd column
        hipDoubleComplex arg_odd_odd     = make_hipDoubleComplex((k + l) / (double) size, 0.0);

        // var for even row and even column
        // 1
        // var for even row and odd column
        hipDoubleComplex var_even_odd    = _cuCexp(hipCmul(hipCmul(var_exp, CU_MPI), arg_even_odd));
        // var for odd row and even column
        hipDoubleComplex var_odd_even    = _cuCexp(hipCmul(hipCmul(var_exp, CU_MPI), arg_odd_even));
        // var for odd row and odd column
        hipDoubleComplex var_odd_odd     = _cuCexp(hipCmul(hipCmul(var_exp, CU_MPI), arg_odd_odd));

        for (int m = 0; m < size/2; m++) {
            for (int n = 0; n < size/2; n++) {
                // calculate e^((-2*pi*i) * (k*m/M + l*n/N))
                hipDoubleComplex arg             = make_hipDoubleComplex(((k*m / (double) size) + (l*n / (double) size)), 0.0);
                hipDoubleComplex exponent        = _cuCexp(hipCmul(hipCmul(var_exp, CU_MPI), arg));
                // make element in matrix to cuDoubleComplex
                hipDoubleComplex el_even_even    = make_hipDoubleComplex(d_mat[(2*m)*size+(2*n)],0.0);
                hipDoubleComplex el_even_odd     = make_hipDoubleComplex(d_mat[(2*m)*size+(2*n+1)],0.0);
                hipDoubleComplex el_odd_even     = make_hipDoubleComplex(d_mat[(2*m+1)*size+(2*n)],0.0);
                hipDoubleComplex el_odd_odd      = make_hipDoubleComplex(d_mat[(2*m+1)*size+(2*n+1)],0.0);
                // add each result to element
                element                         = hipCadd(element, hipCmul(el_even_even, exponent));                          // even row and even column
                element                         = hipCadd(element, hipCmul(el_even_odd, hipCmul(var_even_odd, exponent)));     // even row and odd column
                element                         = hipCadd(element, hipCmul(el_odd_even, hipCmul(var_odd_even, exponent)));     // odd row and even column
                element                         = hipCadd(element, hipCmul(el_odd_odd, hipCmul(var_odd_odd, exponent)));       // odd row and odd column
            }
        } 
        // set element to d_freq
        d_freq[k*size+l] = hipCdiv(element, sizeSquare);   
    }
}


int main(void) {
    struct Matrix     source;
    struct FreqMatrix freq_domain;
    double *d_mat;
    hipDoubleComplex *d_freq;

    readMatrix(&source);
    freq_domain.size = source.size;

    // allocate memory in device
    hipMalloc((void **) &d_mat, source.size * source.size * sizeof(double));
    hipMalloc((void **) &d_freq, source.size * source.size * sizeof(hipDoubleComplex));

    // copy data from host to device
    hipMemcpy(d_mat, source.mat, source.size * source.size * sizeof(double), hipMemcpyHostToDevice);

    // set block size (16 blocks) and grid size (matrix size/block size)
    dim3 block(source.size/BLOCK_SIZE, source.size/BLOCK_SIZE, 1);    
    dim3 grid(BLOCK_SIZE, BLOCK_SIZE, 1);
    // call kernel
    fft<<<block, grid>>>(d_mat, d_freq, source.size);

    // copy data from device to host
    hipMemcpy(freq_domain.mat, d_freq, source.size * source.size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    // free memory in device
    hipFree(d_mat);
    hipFree(d_freq);

    hipDeviceSynchronize();
    
    hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
    hipDoubleComplex size = make_hipDoubleComplex(source.size, 0.0);
    for (int k = 0; k < source.size; k++) {
        for (int l = 0; l < source.size; l++) {
            hipDoubleComplex el = freq_domain.mat[k*freq_domain.size+l];
            printf("(%lf, %lf) ", hipCreal(el), hipCimag(el));
            sum = hipCadd(sum, el);
        }
        printf("\n");
    }
    
    sum = hipCdiv(sum, size);
    printf("Average : (%lf, %lf)\n", hipCreal(sum), hipCimag(sum));

    return 0;
}
