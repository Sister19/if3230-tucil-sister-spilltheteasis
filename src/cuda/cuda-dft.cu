// TUBES SISTER 13520002 CUDA
// cuda-dft.cu
 
// how to run
// > nvcc cuda-dft.cu -o cuda-dft
// > ./cuda-dft
 
// how to measure time
// ex testcase.txt already created, create empty output.txt
// > time ./cuda-dft < 128.txt > output.txt


#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define MAX_N 512
#define BLOCK_SIZE 16
#define CU_MPI make_hipDoubleComplex(M_PI, 0.0)

struct Matrix {
    int    size;
    double mat[MAX_N*MAX_N];
};

struct FreqMatrix {
    int    size;
    hipDoubleComplex mat[MAX_N*MAX_N];
};

void readMatrix(struct Matrix *m) {
    scanf("%d", &(m->size));
    for (int i = 0; i < m->size; i++)
        for (int j = 0; j < m->size; j++)
            scanf("%lf", &(m->mat[i*m->size+j]));
}

__host__ __device__ hipDoubleComplex _cuCexp (hipDoubleComplex arg)
{
   hipDoubleComplex res;
   double s, c;
   double e = exp(arg.x);
   sincos(arg.y, &s, &c);
   res.x = c * e;
   res.y = s * e;
   return res;
}

__global__ void dft(double *d_mat, hipDoubleComplex *d_freq, int size) {
    // get index of thread
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int l = blockIdx.y * blockDim.y + threadIdx.y;

    if (k < size && l < size) {
        // make size to cuDoubleComplex
        hipDoubleComplex sizeSquare      = make_hipDoubleComplex(size*size, 0.0);
        // initialize element in cuDoubleComplex format = 0 + 0i
        hipDoubleComplex element         = make_hipDoubleComplex(0.0, 0.0);  
        // make -2i to cuDoubleComplex
        hipDoubleComplex var_exp         = make_hipDoubleComplex(0.0, -2.0);

        for (int m = 0; m < size; m++) {
            for (int n = 0; n < size; n++) {
                // calculate e^((-2*pi*i) * (k*m/M + l*n/N))
                hipDoubleComplex arg             = make_hipDoubleComplex(((k*m / (double) size) + (l*n / (double) size)), 0.0);
                hipDoubleComplex exponent        = _cuCexp(hipCmul(hipCmul(var_exp, CU_MPI), arg));
                // make element in matrix to cuDoubleComplex
                hipDoubleComplex el              = make_hipDoubleComplex(d_mat[m*size+n],0.0);
                // add result to element
                element                         = hipCadd(element, hipCmul(el, exponent));
            }
        } 
        // set element to d_freq
        d_freq[k*size+l] = hipCdiv(element, sizeSquare);   
    }
}


int main(void) {
    struct Matrix     source;
    struct FreqMatrix freq_domain;
    double *d_mat;
    hipDoubleComplex *d_freq;

    readMatrix(&source);
    freq_domain.size = source.size;

    // allocate memory in device
    hipMalloc((void **) &d_mat, source.size * source.size * sizeof(double));
    hipMalloc((void **) &d_freq, source.size * source.size * sizeof(hipDoubleComplex));

    // copy data from host to device
    hipMemcpy(d_mat, source.mat, source.size * source.size * sizeof(double), hipMemcpyHostToDevice);

    // set block size (16 blocks) and grid size (matrix size/block size)
    dim3 block(source.size/BLOCK_SIZE, source.size/BLOCK_SIZE, 1);    
    dim3 grid(BLOCK_SIZE, BLOCK_SIZE, 1);
    // call kernel
    dft<<<block, grid>>>(d_mat, d_freq, source.size);

    // copy data from device to host
    hipMemcpy(freq_domain.mat, d_freq, source.size * source.size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    // free memory in device
    hipFree(d_mat);
    hipFree(d_freq);

    hipDeviceSynchronize();
    
    hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
    hipDoubleComplex size = make_hipDoubleComplex(source.size, 0.0);
    for (int k = 0; k < source.size; k++) {
        for (int l = 0; l < source.size; l++) {
            hipDoubleComplex el = freq_domain.mat[k*freq_domain.size+l];
            printf("(%lf, %lf) ", hipCreal(el), hipCimag(el));
            sum = hipCadd(sum, el);
        }
        printf("\n");
    }
    
    sum = hipCdiv(sum, size);
    printf("Average : (%lf, %lf)\n", hipCreal(sum), hipCimag(sum));

    return 0;
}
