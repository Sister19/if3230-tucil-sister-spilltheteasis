// TUBES SISTER 13520002 CUDA
// cuda-dft.cu
 
// how to run
// > nvcc cuda-dft.cu -o cuda-dft
// > ./cuda-dft
 
// how to measure time
// ex testcase.txt already created, create empty output.txt
// > time ./cuda-dft < 128.txt > output.txt


#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define MAX_N 512
#define BLOCK_SIZE 16
#define CU_MPI make_hipDoubleComplex(M_PI, 0.0)

struct Matrix {
    int    size;
    double mat[MAX_N*MAX_N];
};

struct FreqMatrix {
    int    size;
    hipDoubleComplex mat[MAX_N*MAX_N];
};

void readMatrix(struct Matrix *m) {
    scanf("%d", &(m->size));
    for (int i = 0; i < m->size; i++)
        for (int j = 0; j < m->size; j++)
            scanf("%lf", &(m->mat[i*m->size+j]));
}

__host__ __device__ hipDoubleComplex _cuCexp (hipDoubleComplex arg)
{
   hipDoubleComplex res;
   double s, c;
   double e = exp(arg.x);
   sincos(arg.y, &s, &c);
   res.x = c * e;
   res.y = s * e;
   return res;
}

__global__ void dft(double *d_mat, hipDoubleComplex *d_freq, int size) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int l = blockIdx.y * blockDim.y + threadIdx.y;

    if (k < size && l < size) {
        hipDoubleComplex sizeSquare      = make_hipDoubleComplex(size*size, 0.0);
        hipDoubleComplex element         = make_hipDoubleComplex(0.0, 0.0);  
        hipDoubleComplex var_exp         = make_hipDoubleComplex(0.0, -2.0);

        for (int m = 0; m < size; m++) {
            for (int n = 0; n < size; n++) {
                hipDoubleComplex arg             = make_hipDoubleComplex(((k*m / (double) size) + (l*n / (double) size)), 0.0);
                hipDoubleComplex exponent        = _cuCexp(hipCmul(hipCmul(var_exp, CU_MPI), arg));
                hipDoubleComplex el              = make_hipDoubleComplex(d_mat[m*size+n],0.0);
                element                         = hipCadd(element, hipCmul(el, exponent));
            }
        } 

        d_freq[k*size+l] = hipCdiv(element, sizeSquare);   
    }
}


int main(void) {
    struct Matrix     source;
    struct FreqMatrix freq_domain;
    double *d_mat;
    hipDoubleComplex *d_freq;

    readMatrix(&source);
    freq_domain.size = source.size;

    hipMalloc((void **) &d_mat, source.size * source.size * sizeof(double));
    hipMalloc((void **) &d_freq, source.size * source.size * sizeof(hipDoubleComplex));

    hipMemcpy(d_mat, source.mat, source.size * source.size * sizeof(double), hipMemcpyHostToDevice);

    dim3 block(source.size/BLOCK_SIZE, source.size/BLOCK_SIZE, 1);    
    dim3 grid(BLOCK_SIZE, BLOCK_SIZE, 1);
    dft<<<block, grid>>>(d_mat, d_freq, source.size);

    hipMemcpy(freq_domain.mat, d_freq, source.size * source.size * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    hipFree(d_mat);
    hipFree(d_freq);
    
    hipDoubleComplex sum = make_hipDoubleComplex(0.0, 0.0);
    hipDoubleComplex size = make_hipDoubleComplex(source.size, 0.0);
    for (int k = 0; k < source.size; k++) {
        for (int l = 0; l < source.size; l++) {
            // printf("%d %d\n", k, l);
            hipDoubleComplex el = freq_domain.mat[k*freq_domain.size+l];
            printf("(%lf, %lf) ", hipCreal(el), hipCimag(el));
            sum = hipCadd(sum, el);
        }
        printf("\n");
    }
    
    sum = hipCdiv(sum, size);
    printf("Average : (%lf, %lf)\n", hipCreal(sum), hipCimag(sum));

    hipDeviceSynchronize();
    return 0;
}
